#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NSTEPS 50
#define VISC 0.1
#define DT 0.1
#define DX 1.0
#define DY 1.0

__global__ void navier_cuda(double *u_new, double *u, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > 0 && i < N-1 && j > 0 && j < N-1) {
        int idx = i * N + j;
        u_new[idx] = u[idx] + VISC * DT * (
            (u[(i+1)*N + j] - 2 * u[idx] + u[(i-1)*N + j]) / (DX*DX) +
            (u[i*N + (j+1)] - 2 * u[idx] + u[i*N + (j-1)]) / (DY*DY)
        );
    }
}

int main(int argc, char **argv) {

    if (argc < 2) {
        printf("Uso: %s <tamanho_malha>\n", argv[0]);
        return 1;
    }
    int N = atoi(argv[1]);

    
    size_t size = N * N * sizeof(double);

    double *h_u = (double *)calloc(N * N, sizeof(double));
    double *h_u_new = (double *)calloc(N * N, sizeof(double));
    h_u[(N / 2) * N + (N / 2)] = 1000.0;

    double *d_u, *d_u_new;
    hipMalloc(&d_u, size);
    hipMalloc(&d_u_new, size);
    hipMemcpy(d_u, h_u, size, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((N + 15) / 16, (N + 15) / 16);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int t = 0; t < NSTEPS; t++)
    {
        navier_cuda<<<grid, block>>>(d_u_new, d_u, N);
        double *tmp = d_u;
        d_u = d_u_new;
        d_u_new = tmp;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    // Log SLURM-friendly:
    printf("Tamanho: %d x %d | Tempo GPU: %.4f ms\n", N, N, ms);

    hipMemcpy(h_u, d_u, size, hipMemcpyDeviceToHost);

    // (Opcional) imprimir soma ou valor central para comparação
    printf("u[%d][%d] = %f\n", N/2, N/2, h_u[(N/2)*N + (N/2)]);

    free(h_u); free(h_u_new);
    hipFree(d_u); hipFree(d_u_new);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
