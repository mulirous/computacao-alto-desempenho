#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NSTEPS 50
#define VISC 0.1
#define DT 0.1
#define DX 1.0
#define DY 1.0

__global__ void navier_cuda(double *u_new, double *u, int N) {
    // Bloco 16x16: shared memory precisa de 18x18 (com borda)
    __shared__ double s_u[34][34];

    int i = blockIdx.y * blockDim.y + threadIdx.y; // linha global
    int j = blockIdx.x * blockDim.x + threadIdx.x; // coluna global
    int li = threadIdx.y + 1; // linha local no shared memory
    int lj = threadIdx.x + 1; // coluna local no shared memory

    // 1. Carregar os dados para o shared memory
    if (i < N && j < N)
        s_u[li][lj] = u[i*N + j];

    // 2. Carregar as bordas do bloco (se na borda do bloco)
    if (threadIdx.y == 0 && i > 0) // topo
        s_u[0][lj] = u[(i-1)*N + j];
    if (threadIdx.y == blockDim.y-1 && i < N-1) // base
        s_u[blockDim.y+1][lj] = u[(i+1)*N + j];
    if (threadIdx.x == 0 && j > 0) // esquerda
        s_u[li][0] = u[i*N + (j-1)];
    if (threadIdx.x == blockDim.x-1 && j < N-1) // direita
        s_u[li][blockDim.x+1] = u[i*N + (j+1)];

    // preenchimento nas quinas/bordas da matriz da memória compartilhada
    // Superior esquerdo
    if (threadIdx.x==0 && threadIdx.y==0 && i>0 && j>0)
        s_u[0][0] = u[(i-1)*N + (j-1)];
    // Superior direito
    if (threadIdx.x==blockDim.x-1 && threadIdx.y==0 && i>0 && j<N-1)
        s_u[0][blockDim.x+1] = u[(i-1)*N + (j+1)];
    // Inferior esquerdo
    if (threadIdx.x==0 && threadIdx.y==blockDim.y-1 && i<N-1 && j>0)
        s_u[blockDim.y+1][0] = u[(i+1)*N + (j-1)];
    // Inferior direito
    if (threadIdx.x==blockDim.x-1 && threadIdx.y==blockDim.y-1 && i<N-1 && j<N-1)
        s_u[blockDim.y+1][blockDim.x+1] = u[(i+1)*N + (j+1)];

    // 3. Sincronizar todas as threads do bloco
    __syncthreads();

    // 4. Fazer o cálculo (apenas para as threads internas, sem bordas)
    if (i > 0 && i < N-1 && j > 0 && j < N-1) {
        int idx = i * N + j;
        u_new[idx] = s_u[li][lj] + VISC * DT * (
            (s_u[li+1][lj] - 2 * s_u[li][lj] + s_u[li-1][lj]) / (DX*DX) +
            (s_u[li][lj+1] - 2 * s_u[li][lj] + s_u[li][lj-1]) / (DY*DY)
        );
    }
}


int main(int argc, char **argv) {

    if (argc < 2) {
        printf("Uso: %s <tamanho_malha>\n", argv[0]);
        return 1;
    }
    int N = atoi(argv[1]);
    size_t size = N*N*sizeof(double);

    double *h_u = (double*)calloc(N*N, sizeof(double));
    double *h_u_new = (double*)calloc(N*N, sizeof(double));
    h_u[(N/2)*N + (N/2)] = 1000.0;

    double *d_u, *d_u_new;
    hipMalloc(&d_u, size);
    hipMalloc(&d_u_new, size);
    hipMemcpy(d_u, h_u, size, hipMemcpyHostToDevice);

    dim3 block(32, 32);
    dim3 grid((N + 31) / 32, (N + 31) / 32);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int t=0; t<NSTEPS; t++) {
        navier_cuda<<<grid, block>>>(d_u_new, d_u, N);
        double *tmp = d_u; d_u = d_u_new; d_u_new = tmp;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    // Log SLURM-friendly:
    printf("Tamanho: %d x %d | Tempo GPU: %.4f ms\n", N, N, ms);

    hipMemcpy(h_u, d_u, size, hipMemcpyDeviceToHost);

    // (Opcional) imprimir soma ou valor central para comparação
    printf("u[%d][%d] = %f\n", N/2, N/2, h_u[(N/2)*N + (N/2)]);

    free(h_u); free(h_u_new);
    hipFree(d_u); hipFree(d_u_new);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
